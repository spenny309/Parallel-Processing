#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>



////////////////////////////////////////////////////////////////////////////////


#include "jpeglib.h"

/*
 * IMAGE DATA FORMATS:
 *
 * The standard input image format is a rectangular array of pixels, with
 * each pixel having the same number of "component" values (color channels).
 * Each pixel row is an array of JSAMPLEs (which typically are unsigned chars).
 * If you are working with color data, then the color values for each pixel
 * must be adjacent in the row; for example, R,G,B,R,G,B,R,G,B,... for 24-bit
 * RGB color.
 */

/* The "frame structure" structure contains an image frame (in RGB or grayscale
 * formats) for passing around the CS338 projects.
 */
typedef struct frame_struct
{
  JSAMPLE *image_buffer;	/* Points to large array of R,G,B-order/grayscale data
                             * Access directly with:
                             *   image_buffer[num_components*pixel + component]
                             */
  JSAMPLE **row_pointers;	/* Points to an array of pointers to the beginning
                             * of each row in the image buffer.  Use to access
                             * the image buffer in a row-wise fashion, with:
                             *   row_pointers[row][num_components*pixel + component]
                             */
  int image_height;		/* Number of rows in image */
  int image_width;		/* Number of columns in image */
  int num_components;	/* Number of components (usually RGB=3 or gray=1) */
} frame_struct_t;
typedef frame_struct_t *frame_ptr;




#define MAXINPUTS 1
#define MAXOUTPUTS 1
frame_ptr input_frames[MAXINPUTS];	/* Pointers to input frames */
frame_ptr output_frames[MAXOUTPUTS];	/* Pointers to output frames */

/* Read/write JPEGs, for program startup & shutdown */
/* YOU SHOULD NOT NEED TO USE THESE AT ALL */
void write_JPEG_file (char * filename, frame_ptr p_info, int quality);
frame_ptr read_JPEG_file (char * filename);

/* Allocate/deallocate frame buffers, USE AS NECESSARY! */
frame_ptr allocate_frame(int height, int width, int num_components);
void destroy_frame(frame_ptr kill_me);

/*
 * write_JPEG_file writes out the contents of an image buffer to a JPEG.
 * A quality level of 2-100 can be provided (default = 75, high quality = ~95,
 * low quality = ~25, utter pixellation = 2).  Note that unlike read_JPEG_file,
 * it does not do any memory allocation on the buffer passed to it.
 */

void write_JPEG_file (char * filename, frame_ptr p_info, int quality)
{
  struct jpeg_compress_struct cinfo;
  struct jpeg_error_mgr jerr;
  FILE * outfile;		/* target file */

  /* Step 1: allocate and initialize JPEG compression object */
  cinfo.err = jpeg_std_error(&jerr);
  jpeg_create_compress(&cinfo);

  /* Step 2: specify data destination (eg, a file) */
  /* Note: steps 2 and 3 can be done in either order. */

  if ((outfile = fopen(filename, "wb")) == NULL) {
    fprintf(stderr, "ERROR: Can't open output file %s\n", filename);
    exit(1);
  }
  jpeg_stdio_dest(&cinfo, outfile);

  /* Step 3: set parameters for compression */

  /* Set basic picture parameters (not optional) */
  cinfo.image_width = p_info->image_width; 	/* image width and height, in pixels */
  cinfo.image_height = p_info->image_height;
  cinfo.input_components = p_info->num_components; /* # of color components per pixel */
  if (p_info->num_components == 3)
    cinfo.in_color_space = JCS_RGB; 	/* colorspace of input image */
  else if (p_info->num_components == 1)
    cinfo.in_color_space = JCS_GRAYSCALE;
  else {
    fprintf(stderr, "ERROR: Non-standard colorspace for compressing!\n");
    exit(1);
  }
  /* Fill in the defaults for everything else, then override quality */
  jpeg_set_defaults(&cinfo);
  jpeg_set_quality(&cinfo, quality, TRUE /* limit to baseline-JPEG values */);

  /* Step 4: Start compressor */
  jpeg_start_compress(&cinfo, TRUE);

  /* Step 5: while (scan lines remain to be written) */
  /*           jpeg_write_scanlines(...); */
  while (cinfo.next_scanline < cinfo.image_height) {
    (void) jpeg_write_scanlines(&cinfo, &(p_info->row_pointers[cinfo.next_scanline]), 1);
  }

  /* Step 6: Finish compression & close output */

  jpeg_finish_compress(&cinfo);
  fclose(outfile);

  /* Step 7: release JPEG compression object */
  jpeg_destroy_compress(&cinfo);
}


/*
 * read_JPEG_file reads the contents of a JPEG into an image buffer, which
 * is automatically allocated after the size of the image is determined.
 * We want to return a frame struct on success, NULL on error.
 */

frame_ptr read_JPEG_file (char * filename)
{
  /* This struct contains the JPEG decompression parameters and pointers to
   * working space (which is allocated as needed by the JPEG library).
   */
  struct jpeg_decompress_struct cinfo;
  struct jpeg_error_mgr jerr;
  FILE * infile;		/* source file */
  frame_ptr p_info;		/* Output frame information */

  //  JSAMPLE *realBuffer;
  //  JSAMPLE **buffer;		/* Output row buffer */
  //  int row_stride;		/* physical row width in output buffer */

  /* Step 1: allocate and initialize JPEG decompression object */
  cinfo.err = jpeg_std_error(&jerr);
  jpeg_create_decompress(&cinfo);

  /* Step 2: open & specify data source (eg, a file) */
  if ((infile = fopen(filename, "rb")) == NULL) {
    fprintf(stderr, "ERROR: Can't open input file %s\n", filename);
    exit(1);
  }
  jpeg_stdio_src(&cinfo, infile);

  /* Step 3: read file parameters with jpeg_read_header() */
  (void) jpeg_read_header(&cinfo, TRUE);

  /* Step 4: use default parameters for decompression */

  /* Step 5: Start decompressor */
  (void) jpeg_start_decompress(&cinfo);

  /* Step X: Create a frame struct & buffers and fill in the blanks */
  fprintf(stderr, "  Opened %s: height = %d, width = %d, c = %d\n",
      filename, cinfo.output_height, cinfo.output_width, cinfo.output_components);
  p_info = allocate_frame(cinfo.output_height, cinfo.output_width, cinfo.output_components);

  /* Step 6: while (scan lines remain to be read) */
  /*           jpeg_read_scanlines(...); */
  while (cinfo.output_scanline < cinfo.output_height) {
    (void) jpeg_read_scanlines(&cinfo, &(p_info->row_pointers[cinfo.output_scanline]), 1);
  }

  /* Step 7: Finish decompression */
  (void) jpeg_finish_decompress(&cinfo);

  /* Step 8: Release JPEG decompression object & file */
  jpeg_destroy_decompress(&cinfo);
  fclose(infile);

  /* At this point you may want to check to see whether any corrupt-data
   * warnings occurred (test whether jerr.pub.num_warnings is nonzero).
   */

  /* And we're done! */
  return p_info;
}


/*
 * allocate/destroy_frame allocate a frame_struct_t and fill in the
 *  blanks appropriately (including allocating the actual frames), and
 *  then destroy them afterwards.
 */

frame_ptr allocate_frame(int height, int width, int num_components)
{
  int row_stride;		/* physical row width in output buffer */
  int i;
  frame_ptr p_info;		/* Output frame information */

  /* JSAMPLEs per row in output buffer */
  row_stride = width * num_components;

  /* Basic struct and information */
  if ((p_info = (frame_struct_t*)malloc(sizeof(frame_struct_t))) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  p_info->image_height = height;
  p_info->image_width = width;
  p_info->num_components = num_components;

  /* Image array and pointers to rows */
  if ((p_info->row_pointers = (JSAMPLE**)malloc(sizeof(JSAMPLE *) * height)) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  if ((p_info->image_buffer = (JSAMPLE*)malloc(sizeof(JSAMPLE) * row_stride * height)) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  for (i=0; i < height; i++)
  	p_info->row_pointers[i] = & (p_info->image_buffer[i * row_stride]);

  /* And send it back! */
  return p_info;
}

void destroy_frame(frame_ptr kill_me)
{
	free(kill_me->image_buffer);
	free(kill_me->row_pointers);
	free(kill_me);
}


void usage()
{
  fprintf(stderr, "ERROR: Need to specify input file and then output file\n");
  exit(1);
}


/* Makes sure values match in the two images*/
void checkResults(frame_ptr f1, frame_ptr f2)
{
  int i, j, k;

  if(f1->image_height != f2->image_height && f1->image_width != f2->image_width
		&& f1->num_components != f2->num_components){
	fprintf(stderr, "Dimensions do not match\n");
	exit(1);
  }

  for (i=0; i < f1->image_height; i++){
    for (j=0; j < f1->image_width; j++){
      for (k=0; k < f1->num_components; k++){
		JSAMPLE j1 = f1->row_pointers[i][(f1->num_components)*j+k];
		JSAMPLE j2 = f2->row_pointers[i][(f2->num_components)*j+k];
		if(j1 != j2){
			fprintf(stderr, "Values do not match at (%d, %d, %d) \n", i, j, k);
			fprintf(stderr, "from %d\n", j1);
			fprintf(stderr, "to %d\n", j2);
			exit(1);
		}
      }
    }
  }

}

void runKernel(frame_ptr result);



/*
 * This is just a helper method. It should call runKernel to set up and
 * invoke the kernel.  It should then also call the uniprocessor version
 * of your blurring code (which does not need to be optimized) and
 * check for correctness of your kernel code.
 */
void
runTest( int argc, char** argv)
{

  frame_ptr from = input_frames[0];
  // Allocate frame for kernel to store its results into
  output_frames[0] = allocate_frame(from->image_height, from->image_width, from->num_components);

  // call kernel
  runKernel(output_frames[0]);

  // TODO : {easy} - invoke uniprocessor version and check results of kernel
                   //to uniprocessor version

}



/**
 * CUDA Kernel Device code
 * This is code for blurring a single pixel
 *
*/
//    //VERSION 1: Uncomment for naive approach.
// __global__ void cs338Blur(unsigned char* from, unsigned char* to, int r,
// 			  int height, int width, int k)
// {
//
//   long col = (blockIdx.x * blockDim.x + threadIdx.x);
//   long row = (blockIdx.y * blockDim.y + threadIdx.y);
//   long this_pixel = (row * width * k) + col * k;
//
// //If current pixel is invalid, do nothing
//   if(col >= width || row >= height) {
//     return;
//   } else {
//     long weight_divisor = 0;
//     int local_weight = 0;
//     // TODO : find solution -- cannot use {k} here; compiler requires constant value.
//     // Wastes space, but still works on greyscale images
//     long blurred_pixels[3] = { 0 };
//     int col_neighbor;
//     int row_neighbor;
//     int curr_dimension;
//     int current_neighbor;
//
//     //For this pixel, find all valid neighbors and calculate weights and values
//     //Bounds check built into for-loop ; less branching this way in cases when row - r or col - r would be very negative
//     for(row_neighbor = ((1 + row - r < 0) ? 0 : (1 + row - r)) ; row_neighbor < row + r && row_neighbor < height ; row_neighbor++){
//       for(col_neighbor = ((1 + col - r < 0) ? 0 : (1 + col - r)) ; col_neighbor < col + r && col_neighbor < width ; col_neighbor++){
//           //Weight adjustment based on abs distance from this_pixel
//           local_weight = (r - abs(row - row_neighbor)) * (r - abs(col - col_neighbor));
//           weight_divisor += local_weight;
//           //current_neighbor = location of R value in RGB
//           current_neighbor = (row_neighbor * width * k) + (col_neighbor * k);
//           for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
//             blurred_pixels[curr_dimension] += from[current_neighbor + curr_dimension] * local_weight;
//           }
//       }
//     }
//
//     //Check for divide by 0 errors
//     if(weight_divisor == 0){
//       return;
//     }
//
//     //Calculate blurred pixel value
//     for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
//       to[this_pixel + curr_dimension] = (unsigned char) (blurred_pixels[curr_dimension] / weight_divisor);
//     }
//
//     return;
//   }
// }

//     //VERSION 2: Uncomment for Block Branching approach
// __global__ void cs338Blur(unsigned char* from, unsigned char* to, int r,
//   int height, int width, int k)
//   {
//     long col = (blockIdx.x * blockDim.x + threadIdx.x);
//     long row = (blockIdx.y * blockDim.y + threadIdx.y);
//     //If current pixel is invalid, do nothing {col && row cann never be < 0, so no need to check}
//     if(col >= width || row >= height) {
//       return;
//     }
//     long this_pixel = (row * width * k) + col * k;
//
//     long weight_divisor = 0;
//     int local_weight = 0;
//     // TODO : find solution -- cannot use {k} here; compiler requires constant value.
//     // Wastes space, but still works on greyscale images
//     long blurred_pixels[3] = { 0 };
//     int col_neighbor;
//     int row_neighbor;
//     int curr_dimension;
//     int current_neighbor;
//     int min_of_height_and_width = min(height, width);
//
//
// // TODO : Ensure this bounds check is accurate on a by-block basis
//     //If we're in an edge case, use boundary checking, else assume we have at least neighbors in each direction
//     if((blockIdx.x * blockDim.x) < r || ((1 + blockIdx.x) * blockDim.x) > min_of_height_and_width || (blockIdx.y * blockDim.y) < r || ((1 + blockIdx.y) * blockDim.y) > min_of_height_and_width){
//       //For this pixel, find all valid neighbors and calculate weights and values
//       //Bounds check built into for-loop ; less branching this way in cases when row - r or col - r would be very negative
//       for(row_neighbor = ((1 + row - r < 0) ? 0 : (1 + row - r)) ; row_neighbor < row + r && row_neighbor < height ; row_neighbor++){
//         for(col_neighbor = ((1 + col - r < 0) ? 0 : (1 + col - r)) ; col_neighbor < col + r && col_neighbor < width ; col_neighbor++){
//           //Weight adjustment based on abs distance from this_pixel
//           local_weight = (r - abs(row - row_neighbor)) * (r - abs(col - col_neighbor));
//           weight_divisor += local_weight;
//           //current_neighbor = location of R value in RGB
//           current_neighbor = (row_neighbor * width * k) + (col_neighbor * k);
//           for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
//             blurred_pixels[curr_dimension] += from[current_neighbor + curr_dimension] * local_weight;
//           }
//         }
//       }
//       //Check for divide by 0 errors {should NEVER trip unless error}
//       if(weight_divisor == 0){
//         return;
//       }
//       //Calculate blurred pixel values
//       for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
//         to[this_pixel + curr_dimension] = (unsigned char) (blurred_pixels[curr_dimension] / weight_divisor);
//       }
//       return;
//     } else {
//       //For this pixel, find all valid neighbors and calculate weights and values
//       //No need for bounds checks in this else case
//       for(row_neighbor = (1 + row - r) ; row_neighbor < row + r ; row_neighbor++){
//         for(col_neighbor = (1 + col - r) ; col_neighbor < col + r ; col_neighbor++){
//           //Weight adjustment based on abs distance from this_pixel
//           local_weight = (r - abs(row - row_neighbor)) * (r - abs(col - col_neighbor));
//           weight_divisor += local_weight;
//           //current_neighbor = location of R value in RGB
//           current_neighbor = (row_neighbor * width * k) + (col_neighbor * k);
//           for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
//             blurred_pixels[curr_dimension] += from[current_neighbor + curr_dimension] * local_weight;
//           }
//         }
//       }
//       //Check for divide by 0 errors {should NEVER trip unless error}
//       if(weight_divisor == 0){
//         return;
//       }
//       //Calculate blurred pixel values
//       for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
//         to[this_pixel + curr_dimension] = (unsigned char) (blurred_pixels[curr_dimension] / weight_divisor);
//       }
//       return;
//     }
//   }

//VERSION 3: Uncomment for Block Branching approach with Pre-calculated values
__global__ void cs338Blur(unsigned char* from, unsigned char* to, int r,
  int height, int width, int k, int * weight_matrix, long pre_calculated_divisor)
  {
    long col = (blockIdx.x * blockDim.x + threadIdx.x);
    long row = (blockIdx.y * blockDim.y + threadIdx.y);
    //If current pixel is invalid, do nothing {col && row cann never be < 0, so no need to check}
    if(col >= width || row >= height) {
      return;
    }
    long this_pixel = (row * width * k) + col * k;
    // TODO : find solution -- cannot use {k} here; compiler requires constant value.
    // Wastes space, but still works on greyscale images
    long blurred_pixels[3] = { 0 };
    int col_neighbor;
    int row_neighbor;
    int curr_dimension;
    int current_neighbor;
    int min_of_height_and_width = min(height, width);

    // TODO : Ensure this bounds check is accurate on a by-block basis
    //If we're in an edge case, use boundary checking, else assume we have r+ neighbors in each direction
    //printf("bIdx: %d\tbDmx: %d\tbIdy: %d\tbDmy: %d\trad : %d\tmhw : %d\n", blockIdx.x, blockDim.x, blockIdx.y, blockDim.y, r, min_of_height_and_width);
    if((blockIdx.x * blockDim.x) < r || ((1 + blockIdx.x) * blockDim.x) > min_of_height_and_width || (blockIdx.y * blockDim.y) < r || ((1 + blockIdx.y) * blockDim.y) > min_of_height_and_width){
      int local_weight;
      long weight_divisor = 0;
      //For this pixel, find all valid neighbors and calculate weights and values
      //Bounds check built into for-loop ; less branching this way in cases when row - r or col - r would be very negative
      for(row_neighbor = (1 + row - r) ; row_neighbor < row + r ; row_neighbor++){
        for(col_neighbor = (1 + col - r) ; col_neighbor < col + r ; col_neighbor++){
          if(row_neighbor > 0 && col_neighbor > 0 && row_neighbor < height && col_neighbor < width){
            //Weight adjustment based on abs distance from this_pixel
            local_weight = (r - abs(row - row_neighbor)) * (r - abs(col - col_neighbor));
            weight_divisor += local_weight;
            //current_neighbor = location of R value in RGB
            current_neighbor = (row_neighbor * width * k) + (col_neighbor * k);
            for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
              blurred_pixels[curr_dimension] += from[current_neighbor + curr_dimension] * local_weight;
            }
          }
        }
      }
      //Check for divide by 0 errors {should NEVER trip unless error}
      if(weight_divisor == 0){
        return;
      }
      //Calculate blurred pixel values
      for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
        to[this_pixel + curr_dimension] = (unsigned char) (blurred_pixels[curr_dimension] / weight_divisor);
      }
      return;
    } else {
      //printf("entering here\n");
      //No need for bounds checks in this else case
      for(row_neighbor = (1 + row - r) ; row_neighbor < row + r ; row_neighbor++){
        for(col_neighbor = (1 + col - r) ; col_neighbor < col + r ; col_neighbor++){
          //current_neighbor = location of R value in RGB
          current_neighbor = (row_neighbor * width * k) + (col_neighbor * k);
          for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
            //use pre-calculated weight matrix to determine weight of current neighbor on blur of current pixel
            blurred_pixels[curr_dimension] += from[current_neighbor + curr_dimension] * weight_matrix[(abs(row - row_neighbor + 1) * r) + abs(col - col_neighbor + 1)];
          }
        }
      }
      //Check for divide by 0 errors {should NEVER trip unless error}
      if(pre_calculated_divisor == 0){
        return;
      }
      //Calculate blurred pixel values
      for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
        to[this_pixel + curr_dimension] = (unsigned char) (blurred_pixels[curr_dimension] / pre_calculated_divisor);
      }
      return;
    }
  }


/**
 * Host main routine
 */
int
main(int argc, char **argv)
{

  if(argc < 3){
    usage();
    exit(1);
  }

  // Load input file
  input_frames[0] = read_JPEG_file(argv[1]);

  // Do the actual work including calling CUDA kernel
  runTest(argc, argv);

  // Write output file
  write_JPEG_file(argv[2], output_frames[0], 75);

  return 0;
}

//********************************************************************************************************************************************

// This sets up GPU device by allocating the required memory and then
// calls the kernel on GPU. (You might choose to add/remove arguments.)
// It's currently set up to use the global variables and write its
// final results into the specified argument.
void
runKernel(frame_ptr result)
{
  frame_ptr from = input_frames[0];
  int picture_height = from->image_height;
  int picture_width = from->image_width;
  int picture_components = from->num_components;
  long array_size_for_memory = picture_width * picture_height * picture_components * sizeof(char);
  int * weight_matrix;
  long pre_calculated_divisor = 0;
  /* TODO : Change radial_param to be a definable val? */
  float radial_param = .05;
  int max_of_width_and_height = (picture_height > picture_width) ? picture_height : picture_width;
  int radius = ceil(max_of_width_and_height * radial_param);

  //Allocate one dimensional array for input picture pixels
  unsigned char *image_as_one_dimensional_array;
  image_as_one_dimensional_array = (unsigned char*)malloc(array_size_for_memory);
  if (image_as_one_dimensional_array == NULL){
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }

  //Allocate one dimensional array for output picture pixels
  JSAMPLE *output_as_one_dimensional_array;
  output_as_one_dimensional_array = (unsigned char*)malloc(array_size_for_memory);
  if (output_as_one_dimensional_array == NULL){
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }

  //Fill input array with picture pixels (row major), and set output array to 200 [light grey]
  int offset = 0;
  printf("looping 1\n");
  for(int i = 0 ; i < picture_height ; i++){
    for(int j = 0 ; j < picture_width ; j++){
      for(int k = 0 ; k < picture_components ; k++){
        offset = (i * picture_width * picture_components) + (j * picture_components) + k;
        image_as_one_dimensional_array[offset] = from->row_pointers[i][(j * picture_components) + k];
        output_as_one_dimensional_array[offset] = 255;
      }
    }
  }
  printf("done loop 1\n");

  //Allocate device memory and transfer input data and output array
  unsigned char* d_image_as_one_dimensional_array;
  unsigned char* d_output_as_one_dimensional_array;
  if (hipMalloc((void **) &d_image_as_one_dimensional_array, array_size_for_memory) != hipSuccess){
    fprintf(stderr, "ERROR: CUDA memory allocation failure\n");
    exit(1);
  }
  if (hipMemcpy(d_image_as_one_dimensional_array, image_as_one_dimensional_array, array_size_for_memory, hipMemcpyHostToDevice) != hipSuccess){
    fprintf(stderr, "1: ERROR: CUDA memory copy failure\n");
    exit(1);
  }

  if (hipMalloc((void **) &d_output_as_one_dimensional_array, array_size_for_memory) != hipSuccess){
    fprintf(stderr, "ERROR: CUDA memory allocation failure\n");
    exit(1);
  }
  if (hipMemcpy(d_output_as_one_dimensional_array, output_as_one_dimensional_array, array_size_for_memory, hipMemcpyHostToDevice) != hipSuccess){
    fprintf(stderr, "2: ERROR: CUDA memory copy failure\n");
    exit(1);
  }

  //Pre-calculate weight divisor matrix
  printf("weight loop\n");
  int weight_matrix_size = sizeof(int) * (radius * radius);
  weight_matrix = (int *)calloc(1, weight_matrix_size);
	for (int i = 0; i < radius; i++){
		for (int j = 0; j < radius; j++){
      weight_matrix[(i*radius) + j] = (radius - i) * (radius - j);
      if(i > 0 && j > 0){
        pre_calculated_divisor += 4 * ((radius - i) * (radius - j));
      } else if (i > 0 || j > 0){
        pre_calculated_divisor += 2 * ((radius - i) * (radius - j));
      } else{
        pre_calculated_divisor += (radius - i) * (radius - j);
      }
		}
	}
  printf("begone loop\n");
  int* d_weight_matrix;
  if (hipMalloc((void **) &d_weight_matrix, weight_matrix_size) != hipSuccess){
    fprintf(stderr, "ERROR: CUDA memory allocation failure\n");
    exit(1);
  }
  if (hipMemcpy(d_weight_matrix, weight_matrix, weight_matrix_size, hipMemcpyHostToDevice) != hipSuccess){
    fprintf(stderr, "3: ERROR: CUDA memory copy failure\n");
    exit(1);
  }

  //Kernel invocation with dimensionality
    /* CURRENT IMPLEMENTATION :
         Wasteful for severely rectangular images, but standard image
         formats are rarely more rectangular than 4:3 or 16:9
         */
         //Add define value for block dimensions
  double block_size = 32.0;
  dim3 dim_grid(ceil(max_of_width_and_height / block_size), ceil(max_of_width_and_height / block_size), 1);
  dim3 dim_block(block_size, block_size, 1);

  printf("calling blur\n");
  cs338Blur<<<dim_grid, dim_block>>>(d_image_as_one_dimensional_array, d_output_as_one_dimensional_array, radius, picture_height, picture_width, picture_components, d_weight_matrix, pre_calculated_divisor);
  //cs338Blur<<<dim_grid, dim_block>>>(d_image_as_one_dimensional_array, d_output_as_one_dimensional_array, radius, picture_height, picture_width, picture_components);

  //Collect results
  if (hipMemcpy(output_as_one_dimensional_array, d_output_as_one_dimensional_array, array_size_for_memory, hipMemcpyDeviceToHost) != hipSuccess){
    fprintf(stderr, "4: ERROR: CUDA memory copy failure\n");
    exit(1);
  }
  printf("begone call\n");

  //Transform into 2D array
  //Fill output image with pixels from hipMemcpy
  for(int i = 0 ; i < picture_height ; i++){
    for(int j = 0 ; j < picture_width ; j++){
      for(int k = 0 ; k < picture_components ; k++){
        offset = (i * picture_width * picture_components) + (j * picture_components) + k;
        result->row_pointers[i][(j * picture_components) + k] = output_as_one_dimensional_array[offset];
      }
    }
  }

  free(image_as_one_dimensional_array);
  free(output_as_one_dimensional_array);
  hipFree(d_image_as_one_dimensional_array);
  hipFree(d_output_as_one_dimensional_array);
}

// Some useful CUDA functions:
// checkCudaErrors is helpful for checking correctness of hipMalloc
// and cudaMemCpy
// You want to use hipEvent_t to get timing information.  Look at
// hipEventCreate, hipEventRecord, hipEventSynchronize,
// hipEventElapsedTime, hipEventDestroy
