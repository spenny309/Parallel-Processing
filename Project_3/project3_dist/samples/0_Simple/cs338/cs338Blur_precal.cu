#include "hip/hip_runtime.h"
// Fully optimized blur code with branch prediction and pre-calculation
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>



////////////////////////////////////////////////////////////////////////////////


#include "jpeglib.h"

#include <string.h>

/*
 * IMAGE DATA FORMATS:
 *
 * The standard input image format is a rectangular array of pixels, with
 * each pixel having the same number of "component" values (color channels).
 * Each pixel row is an array of JSAMPLEs (which typically are unsigned chars).
 * If you are working with color data, then the color values for each pixel
 * must be adjacent in the row; for example, R,G,B,R,G,B,R,G,B,... for 24-bit
 * RGB color.
 */

/* The "frame structure" structure contains an image frame (in RGB or grayscale
 * formats) for passing around the CS338 projects.
 */
typedef struct frame_struct
{
  JSAMPLE *image_buffer;	/* Points to large array of R,G,B-order/grayscale data
                             * Access directly with:
                             *   image_buffer[num_components*pixel + component]
                             */
  JSAMPLE **row_pointers;	/* Points to an array of pointers to the beginning
                             * of each row in the image buffer.  Use to access
                             * the image buffer in a row-wise fashion, with:
                             *   row_pointers[row][num_components*pixel + component]
                             */
  int image_height;		/* Number of rows in image */
  int image_width;		/* Number of columns in image */
  int num_components;	/* Number of components (usually RGB=3 or gray=1) */
} frame_struct_t;
typedef frame_struct_t *frame_ptr;


#ifdef BLOCK
  #define BLOCK_SIZE BLOCK
#else
  //default block_size
  #define BLOCK_SIZE 32.0
#endif

//CHANGE TO UPDATE RADIUS
#define RADIAL_PARAM 0.05f


#define MAXINPUTS 1
#define MAXOUTPUTS 1
frame_ptr input_frames[MAXINPUTS];	/* Pointers to input frames */
frame_ptr output_frames[MAXOUTPUTS];	/* Pointers to output frames */

/* Read/write JPEGs, for program startup & shutdown */
/* YOU SHOULD NOT NEED TO USE THESE AT ALL */
void write_JPEG_file (char * filename, frame_ptr p_info, int quality);
frame_ptr read_JPEG_file (char * filename);

/* Allocate/deallocate frame buffers, USE AS NECESSARY! */
frame_ptr allocate_frame(int height, int width, int num_components);
void destroy_frame(frame_ptr kill_me);

/*
 * write_JPEG_file writes out the contents of an image buffer to a JPEG.
 * A quality level of 2-100 can be provided (default = 75, high quality = ~95,
 * low quality = ~25, utter pixellation = 2).  Note that unlike read_JPEG_file,
 * it does not do any memory allocation on the buffer passed to it.
 */

void write_JPEG_file (char * filename, frame_ptr p_info, int quality)
{
  struct jpeg_compress_struct cinfo;
  struct jpeg_error_mgr jerr;
  FILE * outfile;		/* target file */

  /* Step 1: allocate and initialize JPEG compression object */
  cinfo.err = jpeg_std_error(&jerr);
  jpeg_create_compress(&cinfo);

  /* Step 2: specify data destination (eg, a file) */
  /* Note: steps 2 and 3 can be done in either order. */

  if ((outfile = fopen(filename, "wb")) == NULL) {
    fprintf(stderr, "ERROR: Can't open output file %s\n", filename);
    exit(1);
  }
  jpeg_stdio_dest(&cinfo, outfile);

  /* Step 3: set parameters for compression */

  /* Set basic picture parameters (not optional) */
  cinfo.image_width = p_info->image_width; 	/* image width and height, in pixels */
  cinfo.image_height = p_info->image_height;
  cinfo.input_components = p_info->num_components; /* # of color components per pixel */
  if (p_info->num_components == 3)
    cinfo.in_color_space = JCS_RGB; 	/* colorspace of input image */
  else if (p_info->num_components == 1)
    cinfo.in_color_space = JCS_GRAYSCALE;
  else {
    fprintf(stderr, "ERROR: Non-standard colorspace for compressing!\n");
    exit(1);
  }
  /* Fill in the defaults for everything else, then override quality */
  jpeg_set_defaults(&cinfo);
  jpeg_set_quality(&cinfo, quality, TRUE /* limit to baseline-JPEG values */);

  /* Step 4: Start compressor */
  jpeg_start_compress(&cinfo, TRUE);

  /* Step 5: while (scan lines remain to be written) */
  /*           jpeg_write_scanlines(...); */
  while (cinfo.next_scanline < cinfo.image_height) {
    (void) jpeg_write_scanlines(&cinfo, &(p_info->row_pointers[cinfo.next_scanline]), 1);
  }

  /* Step 6: Finish compression & close output */

  jpeg_finish_compress(&cinfo);
  fclose(outfile);

  /* Step 7: release JPEG compression object */
  jpeg_destroy_compress(&cinfo);
}


/*
 * read_JPEG_file reads the contents of a JPEG into an image buffer, which
 * is automatically allocated after the size of the image is determined.
 * We want to return a frame struct on success, NULL on error.
 */

frame_ptr read_JPEG_file (char * filename)
{
  /* This struct contains the JPEG decompression parameters and pointers to
   * working space (which is allocated as needed by the JPEG library).
   */
  struct jpeg_decompress_struct cinfo;
  struct jpeg_error_mgr jerr;
  FILE * infile;		/* source file */
  frame_ptr p_info;		/* Output frame information */

  //  JSAMPLE *realBuffer;
  //  JSAMPLE **buffer;		/* Output row buffer */
  //  int row_stride;		/* physical row width in output buffer */

  /* Step 1: allocate and initialize JPEG decompression object */
  cinfo.err = jpeg_std_error(&jerr);
  jpeg_create_decompress(&cinfo);

  /* Step 2: open & specify data source (eg, a file) */
  if ((infile = fopen(filename, "rb")) == NULL) {
    fprintf(stderr, "ERROR: Can't open input file %s\n", filename);
    exit(1);
  }
  jpeg_stdio_src(&cinfo, infile);

  /* Step 3: read file parameters with jpeg_read_header() */
  (void) jpeg_read_header(&cinfo, TRUE);

  /* Step 4: use default parameters for decompression */

  /* Step 5: Start decompressor */
  (void) jpeg_start_decompress(&cinfo);

  /* Step X: Create a frame struct & buffers and fill in the blanks */
  fprintf(stderr, "  Opened %s: height = %d, width = %d, c = %d\n",
      filename, cinfo.output_height, cinfo.output_width, cinfo.output_components);
  p_info = allocate_frame(cinfo.output_height, cinfo.output_width, cinfo.output_components);

  /* Step 6: while (scan lines remain to be read) */
  /*           jpeg_read_scanlines(...); */
  while (cinfo.output_scanline < cinfo.output_height) {
    (void) jpeg_read_scanlines(&cinfo, &(p_info->row_pointers[cinfo.output_scanline]), 1);
  }

  /* Step 7: Finish decompression */
  (void) jpeg_finish_decompress(&cinfo);

  /* Step 8: Release JPEG decompression object & file */
  jpeg_destroy_decompress(&cinfo);
  fclose(infile);

  /* At this point you may want to check to see whether any corrupt-data
   * warnings occurred (test whether jerr.pub.num_warnings is nonzero).
   */

  /* And we're done! */
  return p_info;
}


/*
 * allocate/destroy_frame allocate a frame_struct_t and fill in the
 *  blanks appropriately (including allocating the actual frames), and
 *  then destroy them afterwards.
 */

frame_ptr allocate_frame(int height, int width, int num_components)
{
  int row_stride;		/* physical row width in output buffer */
  int i;
  frame_ptr p_info;		/* Output frame information */

  /* JSAMPLEs per row in output buffer */
  row_stride = width * num_components;

  /* Basic struct and information */
  if ((p_info = (frame_struct_t*)malloc(sizeof(frame_struct_t))) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  p_info->image_height = height;
  p_info->image_width = width;
  p_info->num_components = num_components;

  /* Image array and pointers to rows */
  if ((p_info->row_pointers = (JSAMPLE**)malloc(sizeof(JSAMPLE *) * height)) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  if ((p_info->image_buffer = (JSAMPLE*)malloc(sizeof(JSAMPLE) * row_stride * height)) == NULL) {
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  for (i=0; i < height; i++)
  	p_info->row_pointers[i] = & (p_info->image_buffer[i * row_stride]);

  /* And send it back! */
  return p_info;
}

void destroy_frame(frame_ptr kill_me)
{
	free(kill_me->image_buffer);
	free(kill_me->row_pointers);
	free(kill_me);
}


void usage()
{
  fprintf(stderr, "ERROR: Need to specify input file and then output file\n");
  exit(1);
}


/* Makes sure values match in the two images*/
void checkResults(frame_ptr f1, frame_ptr f2)
{
  int i, j, k;

  if(f1->image_height != f2->image_height && f1->image_width != f2->image_width
		&& f1->num_components != f2->num_components){
	fprintf(stderr, "Dimensions do not match\n");
	exit(1);
  }

  for (i=0; i < f1->image_height; i++){
    for (j=0; j < f1->image_width; j++){
      for (k=0; k < f1->num_components; k++){
		JSAMPLE j1 = f1->row_pointers[i][(f1->num_components)*j+k];
		JSAMPLE j2 = f2->row_pointers[i][(f2->num_components)*j+k];
		if(j1 != j2){
			fprintf(stderr, "Values do not match at (%d, %d, %d) \n", i, j, k);
			fprintf(stderr, "from %d\n", j1);
			fprintf(stderr, "to %d\n", j2);
			exit(1);
		}
      }
    }
  }

}

void runKernel(frame_ptr result);



/*
 * This is just a helper method. It should call runKernel to set up and
 * invoke the kernel.  It should then also call the uniprocessor version
 * of your blurring code (which does not need to be optimized) and
 * check for correctness of your kernel code.
 */
void
runTest( int argc, char** argv)
{

  frame_ptr from = input_frames[0];
  // Allocate frame for kernel to store its results into
  output_frames[0] = allocate_frame(from->image_height, from->image_width, from->num_components);

  // call kernel
  runKernel(output_frames[0]);

  // TODO : {easy} - invoke uniprocessor version and check results of kernel
                   //to uniprocessor version

}



/**
 * CUDA Kernel Device code
 * This is code for blurring a single pixel
 *
*/
    //VERSION 3: Block Branching approach with Pre-calculated values
__global__ void cs338Blur(unsigned char* from, unsigned char* to, int r,
  int height, int width, int k, int * weight_matrix, long pre_calculated_divisor)
  {
    long col = (blockIdx.x * blockDim.x + threadIdx.x);
    long row = (blockIdx.y * blockDim.y + threadIdx.y);
    //If current pixel is invalid, do nothing {col && row cann never be < 0, so no need to check}
    if(col >= width || row >= height) {
      return;
    }
    long this_pixel = (row * width * k) + col * k;
    // Wastes space, but still works on greyscale images
    long blurred_pixels[3] = { 0 };
    int col_neighbor;
    int row_neighbor;
    int curr_dimension;
    int current_neighbor;
    int min_of_height_and_width = min(height, width);
    long weight_divisor = pre_calculated_divisor;
    //If we're in an edge case, use boundary checking, else assume we have r+ neighbors in each directions
    if((blockIdx.x * blockDim.x) < r || ((1 + blockIdx.x) * blockDim.x) > width - r || (blockIdx.y * blockDim.y) < r || ((1 + blockIdx.y) * blockDim.y) > height - r){
      int local_weight;
      weight_divisor = 0;
      //For this pixel, find all valid neighbors and calculate weights and values
      for(row_neighbor = (1 + row - r) ; row_neighbor < row + r ; row_neighbor++){
        for(col_neighbor = (1 + col - r) ; col_neighbor < col + r ; col_neighbor++){
          //Check bounds to ensure validity
          if(row_neighbor >= 0 && col_neighbor >= 0 && row_neighbor < height && col_neighbor < width){
            //Weight adjustment based on abs distance from this_pixel
            local_weight = (r - abs(row - row_neighbor)) * (r - abs(col - col_neighbor));
            weight_divisor += local_weight;
            //current_neighbor = location of R value in RGB
            current_neighbor = (row_neighbor * width * k) + (col_neighbor * k);
            for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
              blurred_pixels[curr_dimension] += from[current_neighbor + curr_dimension] * local_weight;
            }
          }
        }
      }
    } else {
      //No need for bounds checks in this else case
      for(row_neighbor = (1 + row - r) ; row_neighbor < row + r ; row_neighbor++){
        for(col_neighbor = (1 + col - r) ; col_neighbor < col + r ; col_neighbor++){
          //current_neighbor = location of R value in RGB
          current_neighbor = (row_neighbor * width * k) + (col_neighbor * k);
          for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
            //use pre-calculated weight matrix to determine weight of current neighbor on blur of current pixel
            blurred_pixels[curr_dimension] += from[current_neighbor + curr_dimension] * weight_matrix[(abs(row - row_neighbor) * r) + abs(col - col_neighbor)];
          }
        }
      }
    }

    //Check for divide by 0 errors {should NEVER trip unless error}
    if(weight_divisor == 0){
      return;
    }
    //Calculate blurred pixel values
    for(curr_dimension = 0 ; curr_dimension < k ; curr_dimension++) {
      to[this_pixel + curr_dimension] = (unsigned char) (blurred_pixels[curr_dimension] / weight_divisor);
    }
    return;
  }

  // UNCOMMENT FOR TEST OUTPUT TXT
  // // Insert this function before main
  // void kelly_write_file(char *fname)
  // {
  //   char *name = (char*)malloc(strlen(fname) + 5);
  //   strcpy(name, fname);
  //   strcat(name, ".out");
  //
  //   FILE *file = fopen(name, "w");
  //   if(file == 0){
  //     printf("Unable to open %s\n", name);
  //   }
  //   else{
  //     int i, j, k;
  //     frame_ptr to;
  //
  //     to = output_frames[0];
  //
  //     for (i=0; i < to->image_height; i++){
  //       for (j=0; j < to->image_width; j++){
  //         for (k=0; k < to->num_components; k++){
  //                   fprintf(file, "%d ", to->row_pointers[i][(to->num_components)*j+k]);
  //         }
  //       }
  //       fprintf(file, "\n");
  //     }
  //   }
  //   fclose(file);
  //   free(name);
  // }

/**
 * Host main routine
 */
int
main(int argc, char **argv)
{

  if(argc < 3){
    usage();
    exit(1);
  }

  // Load input file
  input_frames[0] = read_JPEG_file(argv[1]);

  // Do the actual work including calling CUDA kernel
  runTest(argc, argv);

  // Write output file
  write_JPEG_file(argv[2], output_frames[0], 75);

  // UNCOMMENT FOR TEST OUTPUT TXT
  //frame_ptr compare_to_me = read_JPEG_file(argv[3]);
  //kelly_write_file(argv[3]);

  return 0;
}

//********************************************************************************************************************************************

// This sets up GPU device by allocating the required memory and then
// calls the kernel on GPU. (You might choose to add/remove arguments.)
// It's currently set up to use the global variables and write its
// final results into the specified argument.
void
runKernel(frame_ptr result)
{
  frame_ptr from = input_frames[0];
  int picture_height = from->image_height;
  int picture_width = from->image_width;
  int picture_components = from->num_components;
  long array_size_for_memory = picture_width * picture_height * picture_components * sizeof(char);
  int * weight_matrix;
  long pre_calculated_divisor = 0;
  int max_of_width_and_height = (picture_height > picture_width) ? picture_height : picture_width;
  int radius = ceil(max_of_width_and_height * RADIAL_PARAM);
  hipEvent_t start, stop;
  if (hipEventCreate(&start) != hipSuccess){
    fprintf(stderr, "ERROR: Failed to create CUDA start\n");
    exit(1);
  }
  if (hipEventCreate(&stop) != hipSuccess){
    fprintf(stderr, "ERROR: Failed to create CUDA stop\n");
    exit(1);
  }

  //Allocate one dimensional array for input picture pixels
  unsigned char *image_as_one_dimensional_array;
  image_as_one_dimensional_array = (unsigned char*)malloc(array_size_for_memory);
  if (image_as_one_dimensional_array == NULL){
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }

  //Allocate one dimensional array for output picture pixels
  JSAMPLE *output_as_one_dimensional_array;
  output_as_one_dimensional_array = (unsigned char*)malloc(array_size_for_memory);
  if (output_as_one_dimensional_array == NULL){
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }

  //Fill input array with picture pixels (row major), and set output array to 255 [white]
  int offset = 0;
  for(int i = 0 ; i < picture_height ; i++){
    for(int j = 0 ; j < picture_width ; j++){
      for(int k = 0 ; k < picture_components ; k++){
        offset = (i * picture_width * picture_components) + (j * picture_components) + k;
        image_as_one_dimensional_array[offset] = from->row_pointers[i][(j * picture_components) + k];
        output_as_one_dimensional_array[offset] = 255;
      }
    }
  }

  //Allocate device memory and transfer input data and output array
  unsigned char* d_image_as_one_dimensional_array;
  unsigned char* d_output_as_one_dimensional_array;
  if (hipMalloc((void **) &d_image_as_one_dimensional_array, array_size_for_memory) != hipSuccess){
    fprintf(stderr, "ERROR: CUDA memory allocation failure\n");
    exit(1);
  }
  if (hipMemcpy(d_image_as_one_dimensional_array, image_as_one_dimensional_array, array_size_for_memory, hipMemcpyHostToDevice) != hipSuccess){
    fprintf(stderr, "ERROR: CUDA memory copy failure\n");
    exit(1);
  }

  if (hipMalloc((void **) &d_output_as_one_dimensional_array, array_size_for_memory) != hipSuccess){
    fprintf(stderr, "ERROR: CUDA memory allocation failure\n");
    exit(1);
  }
  if (hipMemcpy(d_output_as_one_dimensional_array, output_as_one_dimensional_array, array_size_for_memory, hipMemcpyHostToDevice) != hipSuccess){
    fprintf(stderr, "ERROR: CUDA memory copy failure\n");
    exit(1);
  }

  //Allocate weight matrix for pre-calculations of inner-pixels
  int weight_matrix_size = sizeof(int) * (radius * radius);
  weight_matrix = (int *)calloc(1, weight_matrix_size);
  if (weight_matrix == NULL){
    fprintf(stderr, "ERROR: Memory allocation failure\n");
    exit(1);
  }
  //Pre-calculate divisor and weight_matrix via simple maths
	for (int i = 0; i < radius; i++){
		for (int j = 0; j < radius; j++){
      weight_matrix[(i*radius) + j] = (radius - i) * (radius - j);
      if (i > 0 && j > 0) { //the 4* covers the 4 quadrant equivalents of i,j
        pre_calculated_divisor += 4 * ((radius - i) * (radius - j));
      } else if (i > 0 || j > 0) { //the 2* covers the 2 axes equivalents of i,j
        pre_calculated_divisor += 2 * ((radius - i) * (radius - j));
      } else { // the 1* covers the one origin at i,j = 0,0
        pre_calculated_divisor += (radius - i) * (radius - j);
      }
		}
	}

  //Create a device copy of weight matrix
  int* d_weight_matrix;
  if (hipMalloc((void **) &d_weight_matrix, weight_matrix_size) != hipSuccess){
    fprintf(stderr, "ERROR: CUDA memory allocation failure\n");
    exit(1);
  }
  if (hipMemcpy(d_weight_matrix, weight_matrix, weight_matrix_size, hipMemcpyHostToDevice) != hipSuccess){
    fprintf(stderr, "ERROR: CUDA memory copy failure\n");
    exit(1);
  }

  //Kernel invocation with dimensionality
  dim3 dim_grid(ceil(picture_width / BLOCK_SIZE), ceil(picture_height / BLOCK_SIZE), 1);
  dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

  hipEventRecord(start);
  cs338Blur<<<dim_grid, dim_block>>>(d_image_as_one_dimensional_array, d_output_as_one_dimensional_array, radius, picture_height, picture_width, picture_components, d_weight_matrix, pre_calculated_divisor);
  hipEventRecord(stop);

  //Collect results with Device to Host memcpy
  if (hipMemcpy(output_as_one_dimensional_array, d_output_as_one_dimensional_array, array_size_for_memory, hipMemcpyDeviceToHost) != hipSuccess){
    fprintf(stderr, "ERROR: CUDA memory copy failure\n");
    exit(1);
  }

  hipEventSynchronize(stop);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  //Transform into 2D array
  //Fill output image with pixels from hipMemcpy
  for(int i = 0 ; i < picture_height ; i++){
    for(int j = 0 ; j < picture_width ; j++){
      for(int k = 0 ; k < picture_components ; k++){
        offset = (i * picture_width * picture_components) + (j * picture_components) + k;
        result->row_pointers[i][(j * picture_components) + k] = output_as_one_dimensional_array[offset];
      }
    }
  }

  printf("Kernal runtime: %10.2f milliseconds\t\tBlock size: %2.1f\n", milliseconds, BLOCK_SIZE);
  free(weight_matrix);
  free(image_as_one_dimensional_array);
  free(output_as_one_dimensional_array);
  hipFree(d_weight_matrix);
  hipFree(d_image_as_one_dimensional_array);
  hipFree(d_output_as_one_dimensional_array);
}
